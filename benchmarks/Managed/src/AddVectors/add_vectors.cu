#include "hip/hip_runtime.h"
// Courtesy of https://devblogs.nvidia.com/parallelforall/unified-memory-cuda-beginners/
// REMOVE ME: Uncommnet the code only upon full implementation or get seg-fault
 
#include <iostream>
#include <math.h>
#include "../../common/util.h"
 
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}
 
int main(int argc, char ** argv)
{
  //uvm-pinning
  // int N = 1<<20;
  size_t N;
  bool pref = false;
  set_envs(&N, sizeof(float), argv[1], argv[2], &pref, argv[3]);
  const int malloc_factor = 2;
  N /= malloc_factor;
  //

  float *x, *y;
 
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
 
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipStream_t stream1;
  hipStream_t stream2;
  hipStream_t stream3;
if (pref) {
  // Prefetch the data to the GPU
  int device = -1;
  hipGetDevice(&device);

  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  hipMemPrefetchAsync(x, N*sizeof(float), device, stream1);
  hipMemPrefetchAsync(y, N*sizeof(float), device, stream2);
}
  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  // int numBlocks = (N + blockSize - 1) / blockSize;
  //uvm-pinning
  int numBlocks;
  if ( (N + blockSize - 1)/blockSize > CUDA_MAX_BLOCK_DIM ) numBlocks = CUDA_MAX_BLOCK_DIM;
  else numBlocks = (N + blockSize - 1) / blockSize;
  
  gpu_timer_set();
if (pref)
  add<<<numBlocks, blockSize, 0, stream3>>>(N, x, y);
else
  add<<<numBlocks, blockSize>>>(N, x, y);
  
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  gpu_timer_pause();
  gpu_timer_record(argc, argv);
 
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  hipFree(x);
  hipFree(y);
 
  return 0;
}

